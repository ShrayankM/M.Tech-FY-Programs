#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define ROWS_A 5000
#define COLS_A 5000

#define ROWS_B 5000
#define COLS_B 5000

#define MAX 128

__global__ void multiplyMatrices(int* a, int *b, int* c, int ra, int ca, int rb, int cb) {

    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    int rno, cno;
    rno = cno = threadId / ra;

    int sum = 0;
    for (int i = 0; i < ca; i++) {
        sum += a[rno * ca + i] * b[i * cb + cno];
    }

    c[rno * cb + (threadId % ra)] = sum;
}

void displayMatrix(int* a, int r, int c) {
    
    for (int i = 0; i < r; i++) {
        for (int j = 0; j < c; j++) {
            printf("%d ", a[i * c + j]);
        }
        printf("\n");
    }
}

int main(int argc, char* argv[]) {

    hipError_t err = hipSuccess;
    hipEvent_t start, stop;
    float elapsed_time_ms;

    int* A = (int*)malloc(ROWS_A * COLS_A * sizeof(int));
    int* B = (int*)malloc(ROWS_B * COLS_B * sizeof(int));

    int* C = (int*)malloc(ROWS_A * COLS_B * sizeof(int));

    //* Initializing Matrix A
    for (int i = 0; i < ROWS_A; i++) {
        for (int j = 0; j < COLS_A; j++) {
            A[i * COLS_A + j] = rand() % MAX + 1;
        }
    }

    //* Initializing Matrix B
    for (int i = 0; i < ROWS_B; i++) {
        for (int j = 0; j < COLS_B; j++) {
            B[i * COLS_B + j] = rand() % MAX + 1;
        }
    }

    int* hA;
    int* hB;

    int* hC;

    if (hipMalloc(&hA, ROWS_A * COLS_A * sizeof(int)) != hipSuccess) {
        printf("Cannot Allocate Memory for A on GPU\n");
        return 0;
    }

    if (hipMalloc(&hB, ROWS_B * COLS_B * sizeof(int)) != hipSuccess) {
        printf("Cannot Allocate Memory for B on GPU\n");
        return 0;
    }

    if (hipMalloc(&hC, ROWS_A * COLS_B * sizeof(int)) != hipSuccess) {
        printf("Cannot Allocate Memory for C on GPU\n");
        return 0;
    }

    if (hipMemcpy(hA, A, ROWS_A * COLS_A * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
        printf("Cannot Data from A on CPU to GPU\n");
        return 0;
    }

    if (hipMemcpy(hB, B, ROWS_B * COLS_B * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
        printf("Cannot Data from B on CPU to GPU\n");
        return 0;
    }

    int threadsPerBlock = 1024;
    int blocks = (ROWS_A * COLS_A + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocks, threadsPerBlock);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    multiplyMatrices<<<blocks, threadsPerBlock>>>(hA, hB, hC, ROWS_A, COLS_A, ROWS_B, COLS_B);
    err = hipGetLastError();
 
    if (err != hipSuccess) {
        printf("Failed to launch multiplyMatrices kernel (error code)!\n");
        return 0;
    }

    if (hipMemcpy(C, hC, ROWS_A * COLS_B * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess) {
        printf("Cannot copy multiplied matrix from GPU to CPU\n");
        return 0;
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);

    printf("The elapsed time is %f seconds\n", elapsed_time_ms / 1000);

    // displayMatrix(C, ROWS_A, COLS_B);

    return 0;
}