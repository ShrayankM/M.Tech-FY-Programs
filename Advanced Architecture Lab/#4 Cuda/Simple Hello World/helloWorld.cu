#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void sayHello()
{
    printf("Basic Info: BlockId = %d,  ThreadId in block =  %d\n", blockIdx.x, threadIdx.x);
    printf("Hello World from the GPU [unique thread no. = %d]\n", blockIdx.x * blockDim.x + threadIdx.x);
}

int main()
{

    sayHello<<<3, 2>>>();
    printf("Hello World from the CPU\n");
    hipDeviceSynchronize();

    return EXIT_SUCCESS;
}
