#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

__global__ void addVectors(int* a, int* b, int N) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < N) {
        a[id] = a[id] + b[id];
    }
}


int main() {

    int N = 4096 * 4096;
    int MAX = 4096;

    hipError_t err = hipSuccess;
    //* Normal vector for initialization
    int* a = (int*)malloc(sizeof(int) * N);
    int* b = (int*)malloc(sizeof(int) * N);

    for (int i = 0; i < N; i++) {
        a[i] = (rand() % MAX) + 1;
        b[i] = (rand() % MAX) + 1;
    }

    printf("Vector Addition of %d elements\n", N);

    // * DS for the GPU
    int* ga;
    int* gb;

    //* Allocating memory for vectors on the GPU
    if (hipMalloc(&ga, sizeof(int) * N) != hipSuccess) {
        printf("Cannot Allocate Memory for A on GPU\n");
        return 0;
    }

    if (hipMalloc(&gb, sizeof(int) * N) != hipSuccess) {
        printf("Cannot Allocate Memory for B on GPU\n");
        return 0;
    }

    //* Copying data contents from CPU to GPU
    if (hipMemcpy(ga, a, sizeof(int) * N, hipMemcpyHostToDevice) != hipSuccess) {
        printf("Cannot Data from A on CPU to GPU\n");
        return 0;
    }

    if (hipMemcpy(gb, b, sizeof(int) * N, hipMemcpyHostToDevice) != hipSuccess) {
        printf("Cannot Data from B on CPU to GPU\n");
        return 0;
    }

    int threadsPerBlock = 1024;
    int blocks =(N + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocks, threadsPerBlock);

    clock_t begin = clock();
    addVectors<<<blocks, threadsPerBlock>>>(ga, gb, N);
    err = hipGetLastError();
 
    if (err != hipSuccess) {
        printf("Failed to launch addVectors kernel (error code)!\n");
        return 0;
    }
    
    if (hipMemcpy(a, ga, sizeof(int) * N, hipMemcpyDeviceToHost) != hipSuccess) {
        printf("Cannot copy added vector from GPU to CPU\n");
        return 0;
    }

    clock_t end = clock();
    printf("The elapsed time is %f seconds\n", (double)(end - begin) / CLOCKS_PER_SEC);

    // for (int i = 0; i < N; i++) 
    //     printf("%d ", a[i]);
    // printf("\n");

    hipFree(ga);
    hipFree(gb);

    free(a);
    free(b);

    return 0;
}