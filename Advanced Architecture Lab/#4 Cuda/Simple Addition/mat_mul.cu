#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#define row1 5
#define col1 5
#define row2 5
#define col2 5

__global__ void matproduct(int *l,int *m, int *n)
{
    int x=blockIdx.x;
    int y=blockIdx.y;
    int k;
  
    n[col2*y+x]=0;
    for(k=0;k<col1;k++)
    {
        n[col2*y+x]=n[col2*y+x]+l[col1*y+k]*m[col2*k+x];
    }
}

int main()
{
    int a[row1][col1];
    int b[row2][col2];
    int c[row1][col2];

    int *d,*e,*f;
    int i,j;
 
    clock_t start, end;
    double cpu_time_used; 
 
    start = clock();    
    for(i=0;i<row1;i++)
    {
        for(j=0;j<col1;j++)
            {
                a[i][j] = 1;
                b[i][j] = 1;
            }
    }    

    hipMalloc((void **)&d,row1*col1*sizeof(int));
    hipMalloc((void **)&e,row2*col2*sizeof(int));
    hipMalloc((void **)&f,row1*col2*sizeof(int));

    hipMemcpy(d,a,row1*col1*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(e,b,row2*col2*sizeof(int),hipMemcpyHostToDevice);

    dim3 grid(col2,row1);       //Here we are defining two dimensional Grid(collection of blocks) structure.
                                  // Syntax is dim3 grid(no. of columns,no. of rows)

    matproduct<<<grid,1>>>(d,e,f);

    hipMemcpy(c,f,row1*col2*sizeof(int),hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    for (int i = 0; i < row1; i++) {
        for (int j = 0; j < col2; j++) {
            printf("%d ", c[i][j]);
        }
        printf("\n");
    }
    
 
     end = clock();
     cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;  
     printf (" time =  %f s\n", cpu_time_used );

    hipFree(d);
    hipFree(e);
    hipFree(f);

    return 0;
}